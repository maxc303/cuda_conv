#include "hip/hip_runtime.h"
#include "helpers.h"

__constant__ float ckernel[81];

__global__ void cuda_padding(float *input, float *input_padded, int pad_size,
                             int width, int height, int width_padded,
                             int height_padded, int channels) {
  int c = blockIdx.z;
  int j = threadIdx.x + blockIdx.x * blockDim.x;
  int i = threadIdx.y + blockIdx.y * blockDim.y;
  int input_idx = i * width * channels + j * channels + c;
  // printf("padding input idx i %d , j %d, c %d \n",i,j,c);

  int output_idx =
      (i + pad_size) * width_padded * channels + (j + pad_size) * channels + c;
  if (i >= height || j >= width) {
    input_padded[output_idx] = 0;
    return;
  }

  input_padded[output_idx] = input[input_idx];
  // printf("padded value ");
}

__global__ void conv_pad_cuda(float *input, float *output, int width,
                              int height, int width_padded, int height_padded,
                              float *kernel, int channels, int k_width,
                              int kernels, int pad_size) {

  int k = blockIdx.z;
  int j = threadIdx.x + blockIdx.x * blockDim.x;
  int i = threadIdx.y + blockIdx.y * blockDim.y;
  int output_idx = i * width * kernels + j * kernels + k;
  int input_idx = 0;
  extern __shared__ float sdata[];

  // Return if out of bound, assign 0 to smem
  if (i >= height || j >= width) {
    // int smem_x = threadIdx.x;
    // int smem_y = threadIdx.y;
    // for (int c = 0; c < channels; c++) {
    //   int smem_index =
    //       (smem_y * (blockDim.x + 2 * k_width) + smem_x) * channels + c;
    //   sdata[smem_index] = 0;

    // }
    return;
  }

  // Copy GMEm to SMEM here
  // Left Overhang
  if (threadIdx.x < k_width) {
    int smem_x = threadIdx.x;
    int smem_y = threadIdx.y + k_width;
    int gmem_x = blockIdx.x * blockDim.x + threadIdx.x - k_width + pad_size;
    int gmem_y = blockIdx.y * blockDim.y + threadIdx.y + pad_size;
    for (int c = 0; c < channels; c++) {
      int gmem_index = gmem_x * channels + gmem_y * width_padded * channels + c;
      int smem_index =
          (smem_y * (blockDim.x + 2 * k_width) + smem_x) * channels + c;
          if(c==0 && blockIdx.x==1 && blockIdx.y==0  ){
            printf("smem x %d , smem_y %d, smem_index %d ,gmem_x %d,gmem_y %d,\n",smem_x,smem_y,smem_index,gmem_x,gmem_y);
           }
     // sdata[smem_index] = input[gmem_index];
  
    }
  }
  // Top Overhang
  if (threadIdx.y < k_width) {
    int smem_x = threadIdx.x + k_width;
    int smem_y = threadIdx.y;
    int gmem_x = blockIdx.x * blockDim.x + threadIdx.x + pad_size;
    int gmem_y = blockIdx.y * blockDim.y + threadIdx.y - k_width + pad_size;
    for (int c = 0; c < channels; c++) {
      int gmem_index = gmem_x * channels + gmem_y * width_padded * channels + c;
      int smem_index =
          (smem_y * (blockDim.x + 2 * k_width) + smem_x) * channels + c;

      sdata[smem_index] = input[gmem_index];
    }
  }
  // Top Left
  if (threadIdx.y < k_width && threadIdx.x < k_width) {
    int smem_x = threadIdx.x;
    int smem_y = threadIdx.y;
    int gmem_x = blockIdx.x * blockDim.x + threadIdx.x - k_width + pad_size;
    int gmem_y = blockIdx.y * blockDim.y + threadIdx.y - k_width + pad_size;
    for (int c = 0; c < channels; c++) {
      int gmem_index = gmem_x * channels + gmem_y * width_padded * channels + c;
      int smem_index =
          (smem_y * (blockDim.x + 2 * k_width) + smem_x) * channels + c;

      sdata[smem_index] = input[gmem_index];
    }
  }
  // Top Right
  if (threadIdx.y < k_width && threadIdx.x >= blockDim.x - k_width) {
    int smem_x = threadIdx.x + 2 * k_width;
    int smem_y = threadIdx.y;
    int gmem_x = blockIdx.x * blockDim.x + threadIdx.x + k_width + pad_size;
    int gmem_y = blockIdx.y * blockDim.y + threadIdx.y - k_width + pad_size;
    for (int c = 0; c < channels; c++) {
      int gmem_index = gmem_x * channels + gmem_y * width_padded * channels + c;
      int smem_index =
          (smem_y * (blockDim.x + 2 * k_width) + smem_x) * channels + c;
      sdata[smem_index] = input[gmem_index];
    }
  }
    // Bottom Left
    if (threadIdx.x < k_width && threadIdx.y >= blockDim.y - k_width) {
      int smem_x = threadIdx.x;
      int smem_y = threadIdx.y + 2 * k_width;
      int gmem_x = blockIdx.x * blockDim.x + threadIdx.x - k_width + pad_size;
      int gmem_y = blockIdx.y * blockDim.y + threadIdx.y + k_width + pad_size;
      for (int c = 0; c < channels; c++) {
        int gmem_index =
            gmem_x * channels + gmem_y * width_padded * channels + c;
        int smem_index =
            (smem_y * (blockDim.x + 2 * k_width) + smem_x) * channels + c;

        sdata[smem_index] = input[gmem_index];
      }
    }
    // Bottom
    if (threadIdx.y >= blockDim.y - k_width) {
      int smem_x = threadIdx.x + k_width;
      int smem_y = threadIdx.y + 2 * k_width;
      int gmem_x = blockIdx.x * blockDim.x + threadIdx.x + pad_size;
      int gmem_y = blockIdx.y * blockDim.y + threadIdx.y + k_width + pad_size;
      for (int c = 0; c < channels; c++) {
        int gmem_index =
            gmem_x * channels + gmem_y * width_padded * channels + c;
        int smem_index =
            (smem_y * (blockDim.x + 2 * k_width) + smem_x) * channels + c;
        sdata[smem_index] = input[gmem_index];
      }
    }
    // Right
    if (threadIdx.x >= blockDim.x - k_width) {
      int smem_x = threadIdx.x + 2 * k_width;
      int smem_y = threadIdx.y + k_width;
      int gmem_x = blockIdx.x * blockDim.x + threadIdx.x + k_width + pad_size;
      int gmem_y = blockIdx.y * blockDim.y + threadIdx.y + pad_size;
      for (int c = 0; c < channels; c++) {
        int gmem_index =
            gmem_x * channels + gmem_y * width_padded * channels + c;
        int smem_index =
            (smem_y * (blockDim.x + 2 * k_width) + smem_x) * channels + c;
        sdata[smem_index] = input[gmem_index];
      }
    }

    // Bottom Right
    if (threadIdx.x >= blockDim.x - k_width &&
        threadIdx.y >= blockDim.y - k_width) {
      int smem_x = threadIdx.x + 2 * k_width;
      int smem_y = threadIdx.y + 2 * k_width;
      int gmem_x = blockIdx.x * blockDim.x + threadIdx.x + k_width + pad_size;
      int gmem_y = blockIdx.y * blockDim.y + threadIdx.y + k_width + pad_size;
      for (int c = 0; c < channels; c++) {
        int gmem_index =
            gmem_x * channels + gmem_y * width_padded * channels + c;
        int smem_index =
            (smem_y * (blockDim.x + 2 * k_width) + smem_x) * channels + c;
            (gmem_x >= width || gmem_y >= height) ? 0 : input[gmem_index];
          }
    }

    // Copy the block data
    int smem_x = threadIdx.x + k_width;
    int smem_y = threadIdx.y + k_width;
    int gmem_x = blockIdx.x * blockDim.x + threadIdx.x + pad_size;
    int gmem_y = blockIdx.y * blockDim.y + threadIdx.y + pad_size;
    for (int c = 0; c < channels; c++) {
      int gmem_index = gmem_x * channels + gmem_y * width_padded * channels + c;
      int smem_index =
          (smem_y * (blockDim.x + 2 * k_width) + smem_x) * channels + c;
      sdata[smem_index] = input[gmem_index];
    }

    __syncthreads();

    // float tmp_output = 0;

    for (int c = 0; c < channels; c++) {
      for (int k_i = 0; k_i <= 2 * k_width; k_i++) {
        for (int k_j = 0; k_j <= 2 * k_width; k_j++) {

          smem_x = threadIdx.x + k_j;
          smem_y = threadIdx.y + k_i;
          int smem_index = c + smem_x * channels +
                           smem_y * channels * (blockDim.x + 2 * k_width);
                    
                           
          int kernel_index =
              k * channels * (2 * k_width + 1) * (2 * k_width + 1) +
              c * (2 * k_width + 1) * (2 * k_width + 1) +
              k_i * (2 * k_width + 1) + k_j;
          tmp_output += sdata[smem_index] * kernel[kernel_index];
        }
      }
    }

    float tmp_output = 0;

    for (int c = 0; c < channels; c++) {
      for (int k_i = -k_width; k_i <= k_width; k_i++) {
        for (int k_j = -k_width; k_j <= k_width; k_j++) {
          input_idx = (i + k_i + pad_size) * width_padded * channels +
                      (j + k_j + pad_size) * channels + c;
          int kernel_index =
              k * channels * (2 * k_width + 1) * (2 * k_width + 1) +
              c * (2 * k_width + 1) * (2 * k_width + 1) +
              (k_i + k_width) * (2 * k_width + 1) + k_j + k_width;
              tmp_output += input[input_idx] * ckernel[kernel_index];
          // h_kernel[k][c][k_i + k_width][k_j + k_width];
        }
      }
    }
    output[output_idx] = tmp_output;

    return;
  }

  int main(int argc, char *argv[]) {
    char *outputfile = (char *)"cuda_cmem_pad.png";
    // Check input image name
    if (argc < 2) {
      std::cout << "No file input" << std::endl;
      return 0;
    }
    //
    // Check if the filename is valid
    char *filename = argv[1];
    std::cout << argv[1] << std::endl;
    // Load Image
    cv::Mat image;
    image = load_image(filename);
    if (image.empty()) {
      std::cout << "File not exist" << std::endl;
      return 0;
    }

    //==================================
    // Define I/O sizes
    //==================================
    int padding = 1;
    int channels = 3;
    int height = image.rows;
    int width = image.cols;
    int kernels = 3;

    std::cout << "Image dims (HxW)is " << height << "x" << width << std::endl;
    int height_padded = height + 2 * padding;
    int width_padded = width + 2 * padding;
    int input_bytes = channels * height * width * sizeof(float);
    int output_bytes = channels * height * width * sizeof(float);
    std::cout << "Padded dims is " << height_padded << "x" << width_padded
              << std::endl;
    int padded_bytes = width_padded * height_padded * channels * sizeof(float);
    float *h_input = (float *)image.data;

    // float *h_output = new float[output_bytes];
    float *h_output;
    h_output = (float *)malloc(output_bytes);
    float *d_input;
    float *d_pad_input;
    float *d_output;
    hipMalloc((void **)&d_input, input_bytes);
    hipMalloc((void **)&d_pad_input, padded_bytes);
    hipMalloc((void **)&d_output, output_bytes);
    hipMemcpy(d_input, h_input, input_bytes, hipMemcpyHostToDevice);

    // invoke Kernel
    int bx = 32;
    int by = 32;
    dim3 block(bx, by); // you will want to configure this
    dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y,
              3);
    printf("Grid : {%d, %d, %d} blocks. Blocks : {%d, %d} threads.\n", grid.x,
           grid.y, grid.z, block.x, block.y);

    //==================================
    // Define Kernel data
    //==================================
    // Mystery kernel

    const float kernel_template[3][3] = {{1, 1, 1}, {1, -8, 1}, {1, 1, 1}};
    float *d_kernel;
    float h_kernel[3][3][3][3];
    int kernel_bytes = 3 * 3 * 3 * 3 * sizeof(float);
    for (int kernel = 0; kernel < 3; ++kernel) {
      for (int channel = 0; channel < 3; ++channel) {
        for (int row = 0; row < 3; ++row) {
          for (int column = 0; column < 3; ++column) {
            h_kernel[kernel][channel][row][column] =
                kernel_template[row][column];
          }
        }
      }
    }

    // //Copy kernel to global mem
    // hipMalloc( (void **) &d_kernel, kernel_bytes ) ;
    // hipMemcpy( d_kernel, h_kernel, kernel_bytes, hipMemcpyHostToDevice);
    // Copy kernek to cmem
    hipMemcpyToSymbol(HIP_SYMBOL(ckernel), &h_kernel, kernel_bytes);

    int k_size = 3;
    int k_width = (k_size - 1) / 2;
    int smem_size =
        (bx + 2 * k_width) * (by + 2 * k_width) * channels * sizeof(float);
    printf("SMEM size is %d \n", (bx + 2 * k_width) * (by + 2 * k_width));
    //==================================
    // CPU Convolution
    //==================================
    printf("Start conv\n");
    double timeStampA = getTimeStamp();

    cuda_padding<<<grid, block>>>(d_input, d_pad_input, padding, width, height,
                                  width_padded, height_padded, channels);

    hipDeviceSynchronize();
    double timeStampC = getTimeStamp();
    conv_pad_cuda<<<grid, block, smem_size>>>(
        d_pad_input, d_output, width, height, width_padded, height_padded,
        ckernel, 3, k_width, kernels, padding);
    hipDeviceSynchronize();
    double timeStampB = getTimeStamp();

    hipMemcpy(h_output, d_output, input_bytes, hipMemcpyDeviceToHost);

    //==================================
    // Collect data
    //==================================

    // Print result
    std::cout << "Total convolution time: " << timeStampB - timeStampA
              << std::endl;
    std::cout << "Padding time: " << timeStampC - timeStampA << std::endl;
    std::cout << "Save Output to " << outputfile << std::endl;
    save_image(outputfile, h_output, height, width);

    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_pad_input);
    hipFree(d_kernel);
    hipDeviceReset();

    return 0;
  }