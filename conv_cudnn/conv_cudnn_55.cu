#include <hipDNN.h>
#include <iostream>
#include <opencv2/opencv.hpp>
#include <sys/time.h>
// Check cudnn
#define checkCUDNN(expression)                                                 \
  {                                                                            \
    hipdnnStatus_t status = (expression);                                       \
    if (status != HIPDNN_STATUS_SUCCESS) {                                      \
      std::cerr << "Error on line " << __LINE__ << ": "                        \
                << hipdnnGetErrorString(status) << std::endl;                   \
      std::exit(EXIT_FAILURE);                                                 \
    }                                                                          \
  }

// Get time function
double getTimeStamp() {
  struct timeval tv;
  gettimeofday(&tv, NULL);
  return (double)tv.tv_usec / 1000000 + tv.tv_sec;
}

// Load Image function
cv::Mat load_image(const char *image_path) {
  cv::Mat image = cv::imread(image_path, CV_LOAD_IMAGE_COLOR);
  image.convertTo(image, CV_32FC3);
  cv::normalize(image, image, 0, 1, cv::NORM_MINMAX);
  return image;
}

// Save Image function
void save_image(const char *output_filename, float *buffer, int height,
                int width) {
  cv::Mat output_image(height, width, CV_32FC3, buffer);
  // Make negative values zero.
  cv::threshold(output_image, output_image,
                /*threshold=*/0,
                /*maxval=*/0, cv::THRESH_TOZERO);
  cv::normalize(output_image, output_image, 0.0, 255.0, cv::NORM_MINMAX);
  output_image.convertTo(output_image, CV_8UC3);

  cv::imwrite(output_filename, output_image);
}

int main(int argc, char *argv[]) {
  // Init cudnn
  hipDeviceReset();

  hipdnnHandle_t cudnn;
  hipdnnCreate(&cudnn);

  char *outputfile = (char *)"cudnn_out_55.png";
  // Check input image name
  if (argc < 2) {
    std::cout << "No file input" << std::endl;
    return 0;
  }
  //
  // Check if the filename is valid
  char *filename = argv[1];
  std::cout << argv[1] << std::endl;
  // Load Image
  cv::Mat image;
  image = load_image(filename);
  if (image.empty()) {
    std::cout << "File not exist" << std::endl;
    return 0;
  }

  // Input Descriptor
  hipdnnTensorDescriptor_t input_descriptor;
  checkCUDNN(hipdnnCreateTensorDescriptor(&input_descriptor));
  checkCUDNN(hipdnnSetTensor4dDescriptor(input_descriptor,
                                        /*format=*/HIPDNN_TENSOR_NHWC,
                                        /*dataType=*/HIPDNN_DATA_FLOAT,
                                        /*batch_size=*/1,
                                        /*channels=*/3,
                                        /*image_height=*/image.rows,
                                        /*image_width=*/image.cols));

  hipdnnTensorDescriptor_t output_descriptor;
  checkCUDNN(hipdnnCreateTensorDescriptor(&output_descriptor));
  checkCUDNN(hipdnnSetTensor4dDescriptor(output_descriptor,
                                        /*format=*/HIPDNN_TENSOR_NHWC,
                                        /*dataType=*/HIPDNN_DATA_FLOAT,
                                        /*batch_size=*/1,
                                        /*channels=*/3,
                                        /*image_height=*/image.rows,
                                        /*image_width=*/image.cols));

  hipdnnFilterDescriptor_t kernel_descriptor;
  checkCUDNN(hipdnnCreateFilterDescriptor(&kernel_descriptor));
  checkCUDNN(hipdnnSetFilter4dDescriptor(kernel_descriptor,
                                        /*dataType=*/HIPDNN_DATA_FLOAT,
                                        /*format=*/HIPDNN_TENSOR_NCHW,
                                        /*out_channels=*/3,
                                        /*in_channels=*/3,
                                        /*kernel_height=*/5,
                                        /*kernel_width=*/5));

  hipdnnConvolutionDescriptor_t convolution_descriptor;
  checkCUDNN(hipdnnCreateConvolutionDescriptor(&convolution_descriptor));
  checkCUDNN(hipdnnSetConvolution2dDescriptor(convolution_descriptor,
                                             /*pad_height=*/2,
                                             /*pad_width=*/2,
                                             /*vertical_stride=*/1,
                                             /*horizontal_stride=*/1,
                                             /*dilation_height=*/1,
                                             /*dilation_width=*/1,
                                             /*mode=*/HIPDNN_CROSS_CORRELATION,
                                             /*computeType=*/HIPDNN_DATA_FLOAT));

  hipdnnConvolutionFwdAlgo_t convolution_algorithm;
  checkCUDNN(hipdnnGetConvolutionForwardAlgorithm(
      cudnn, input_descriptor, kernel_descriptor, convolution_descriptor,
      output_descriptor, HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
      /*memoryLimitInBytes=*/0, &convolution_algorithm));

  size_t workspace_bytes = 0;

  checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(
      cudnn, input_descriptor, kernel_descriptor, convolution_descriptor,
      output_descriptor, convolution_algorithm, &workspace_bytes));
  std::cerr << "Workspace size: " << (workspace_bytes / 1048576.0) << "MB"
            << std::endl;

  void *d_workspace;
  hipMalloc(&d_workspace, workspace_bytes);
  std::cout << "allocate workspace" << std::endl;
  int batch_size;
  int channels;
  int height;
  int width;

  hipdnnGetConvolution2dForwardOutputDim(
      convolution_descriptor, input_descriptor, kernel_descriptor, &batch_size,
      &channels, &height, &width);

  int image_bytes = batch_size * channels * height * width * sizeof(float);

  float *d_input;
  hipMalloc(&d_input, image_bytes);
  hipMemcpy(d_input, image.ptr<float>(0), image_bytes, hipMemcpyHostToDevice);

  float *d_output;
  hipMalloc(&d_output, image_bytes);
  hipMemset(d_output, 0, image_bytes);

  std::cout << "Height and width:" << height << " x " << width << std::endl;
  // Mystery kernel
  const float kernel_template[5][5] = {{1, 1, 1, 1, 1},
                                       {1, 4, 4, 4, 1},
                                       {1, 4, 12, 4, 1},
                                       {1, 4, 4, 4, 1},
                                       {1, 1, 1, 1, 1}};

  float h_kernel[3][3][5][5];
  for (int kernel = 0; kernel < 3; ++kernel) {
    for (int channel = 0; channel < 3; ++channel) {
      for (int row = 0; row < 5; ++row) {
        for (int column = 0; column < 5; ++column) {
          h_kernel[kernel][channel][row][column] = kernel_template[row][column];
        }
      }
    }
  }

  float *d_kernel;
  hipMalloc(&d_kernel, sizeof(h_kernel));
  hipMemcpy(d_kernel, h_kernel, sizeof(h_kernel), hipMemcpyHostToDevice);
  const float alpha = 1, beta = 0;

  std::cout << "Start conv" << std::endl;
  double timeStampA = getTimeStamp();
  checkCUDNN(hipdnnConvolutionForward(
      cudnn, &alpha, input_descriptor, d_input, kernel_descriptor, d_kernel,
      convolution_descriptor, convolution_algorithm, d_workspace,
      workspace_bytes, &beta, output_descriptor, d_output));

  hipDeviceSynchronize();
  double timeStampB = getTimeStamp();
  float *h_output = new float[image_bytes];
  hipMemcpy(h_output, d_output, image_bytes, hipMemcpyDeviceToHost);

  // Print result
  std::cout << "Total convolution time: " << timeStampB - timeStampA
            << std::endl;
  std::cout << "Save Output to " << outputfile << std::endl;
  save_image(outputfile, h_output, height, width);

  // Delete
  delete[] h_output;
  hipFree(d_kernel);
  hipFree(d_input);
  hipFree(d_output);
  hipFree(d_workspace);

  hipdnnDestroyTensorDescriptor(input_descriptor);
  hipdnnDestroyTensorDescriptor(output_descriptor);
  hipdnnDestroyFilterDescriptor(kernel_descriptor);
  hipdnnDestroyConvolutionDescriptor(convolution_descriptor);

  hipdnnDestroy(cudnn);
}
